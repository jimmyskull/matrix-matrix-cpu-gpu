#include "cublas.hpp"

#include <cstdio>

#include <stdexcept>
#include <iostream>

#include <hipblas.h>

namespace core {

struct CUBLASGPUInternals {
  float* dev_a;
  float* dev_b;
  float* dev_c;
  hipblasHandle_t handle;
};

CUBLAS::CUBLAS() : Algorithm() {
  internal = new CUBLASGPUInternals;
}

CUBLAS::~CUBLAS() {
  delete internal;
}

void CUBLAS::Init(const reader::Entry& entry) {
  Algorithm::Init(entry);
  std::size_t N = static_cast<std::size_t>(entry.vertices);
  std::size_t N2 = N * N;

  int status;

  status = hipblasCreate(&internal->handle);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
  }

  /* Allocate device memory for the matrices */
  if (hipMalloc((void **)&internal->dev_a, N2 * sizeof(internal->dev_a[0])) != hipSuccess) {
      fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
  }

  if (hipMalloc((void **)&internal->dev_b, N2 * sizeof(internal->dev_b[0])) != hipSuccess) {
      fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
  }

  if (hipMalloc((void **)&internal->dev_c, N2 * sizeof(internal->dev_c[0])) != hipSuccess) {
    fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
  }

}

void CUBLAS::Finalize() {
  if (hipFree(internal->dev_a) != hipSuccess) {
      fprintf(stderr, "!!!! memory free error (A)\n");
  }

  if (hipFree(internal->dev_b) != hipSuccess) {
      fprintf(stderr, "!!!! memory free error (B)\n");
  }

  if (hipFree(internal->dev_c) != hipSuccess) {
      fprintf(stderr, "!!!! memory free error (C)\n");
  }

  hipblasDestroy(internal->handle);
}

void CUBLAS::Compute() {
  std::size_t N = static_cast<std::size_t>(entry.vertices);
  std::size_t N2 = N * N;
  hipblasStatus_t status;
  float alpha = 1.0f;
  float beta = 0.0f;

  status = hipblasSetVector(N2, sizeof(A[0]), A, 1, internal->dev_a, 1);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! device access error (write A)\n");
  }

  status = hipblasSetVector(N2, sizeof(B[0]), B, 1, internal->dev_b, 1);

  if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "!!!! device access error (write B)\n");
  }

  status = hipblasSetVector(N2, sizeof(C[0]), C, 1, internal->dev_c, 1);

  if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "!!!! device access error (write C)\n");
  }

  status = hipblasSgemm(internal->handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    1, N, N,
    &alpha, internal->dev_a, 1, internal->dev_b,
    N, &beta, internal->dev_c, 1);

  if (status != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "!!!! kernel execution error.\n");
  }
}

} // namespace core


#include "cusparse.hpp"

#include <cstdio>
#include <cstdlib>

#include <hipsparse.h>

#include "cutil.hpp"

namespace core {

struct COOSparseMatrixHost {
  COOSparseMatrixHost();

  ~COOSparseMatrixHost();

  // Allocate space according to |nnz|
  void Allocate();

  float* values;
  int* row_indices;
  int* column_indices;
  int nnz;
  int rows;
};

struct COOSparseMatrixDevice {
  COOSparseMatrixDevice();

  ~COOSparseMatrixDevice();

  void CopyFrom(const COOSparseMatrixHost& host, hipsparseHandle_t handle);

  hipsparseMatDescr_t mat;
  float* values;
  int* row_indices;
  int* column_indices;
  int* csr_row_ptr;
  int nnz;
  int rows;

private:
  void Allocate();
};

template<typename SpMatType>
struct SparseDataStructures {
  SpMatType A;
  SpMatType B;
  SpMatType C;
};

struct CUSPARSEInternals {
  hipsparseHandle_t handle;
  SparseDataStructures<COOSparseMatrixHost> host;
  SparseDataStructures<COOSparseMatrixDevice> dev;
};

COOSparseMatrixHost::COOSparseMatrixHost() {
  values = NULL;
  row_indices = NULL;
  column_indices = NULL;
  nnz = 0;
  rows = 0;
}

COOSparseMatrixHost::~COOSparseMatrixHost() {
  if (values) {
    delete[] values;
    values = NULL;
  }
  if (row_indices) {
    delete[] row_indices;
    row_indices = NULL;
  }
  if (column_indices) {
    delete[] column_indices;
    column_indices = NULL;
  }
}

void COOSparseMatrixHost::Allocate() {
  if (nnz == 0) {
    fprintf(stderr, "COOSparseMatrixHost Warning: nnz is zero.\n");
    return;
  }
  values = new float[nnz];
  row_indices = new int[nnz];
  column_indices = new int[nnz];
}

COOSparseMatrixDevice::COOSparseMatrixDevice() {
  hipsparseStatus_t stat;

  stat = hipsparseCreateMatDescr(&mat);
  cusparseSafeCall(stat);

  stat = hipsparseSetMatType(mat, HIPSPARSE_MATRIX_TYPE_GENERAL);
  cusparseSafeCall(stat);

  stat = hipsparseSetMatIndexBase(mat, HIPSPARSE_INDEX_BASE_ZERO);
  cusparseSafeCall(stat);

  row_indices = NULL;
  column_indices = NULL;
  values = NULL;
  csr_row_ptr = NULL;
  nnz = 0;
  rows = 0;
}

COOSparseMatrixDevice::~COOSparseMatrixDevice() {
  hipError_t custat;
  hipsparseStatus_t stat;

  if (values != NULL) {
    custat = hipFree(values);
    cudaSafeCall(custat);
    values = NULL;
  }

  if (row_indices != NULL) {
    custat = hipFree(row_indices);
    cudaSafeCall(custat);
    row_indices = NULL;
  }

  if (column_indices != NULL) {
    custat = hipFree(column_indices);
    cudaSafeCall(custat);
    column_indices = NULL;
  }

  if (csr_row_ptr != NULL) {
    custat = hipFree(csr_row_ptr);
    cudaSafeCall(custat);
    csr_row_ptr = NULL;
  }

  stat = hipsparseDestroyMatDescr(mat);
  cusparseSafeCall(stat);
}

void COOSparseMatrixDevice::Allocate() {
  if (nnz == 0) {
    fprintf(stderr, "COOSparseMatrixDevice Warning: nnz is zero.\n");
    return;
  }
  hipError_t status;

  status = hipMalloc((void**)&values, nnz * sizeof(values[0]));
  cudaSafeCall(status);

  status = hipMalloc((void**)&row_indices, nnz * sizeof(row_indices[0]));
  cudaSafeCall(status);

  status = hipMalloc((void**)&column_indices, nnz * sizeof(column_indices[0]));
  cudaSafeCall(status);

  status = hipMalloc((void**)&csr_row_ptr, (rows + 1) * sizeof(csr_row_ptr[0]));
  cudaSafeCall(status);
}

void COOSparseMatrixDevice::CopyFrom(const COOSparseMatrixHost& host,
  hipsparseHandle_t handle) {
  hipError_t stat;
  hipsparseStatus_t status;

  rows = host.rows;
  nnz = host.nnz;
  Allocate();

  stat = hipMemcpy(row_indices, host.row_indices,
    static_cast<std::size_t>(nnz * sizeof(host.row_indices[0])),
                         hipMemcpyHostToDevice);
  cudaSafeCall(stat);

  stat = hipMemcpy(column_indices, host.column_indices,
    static_cast<std::size_t>(nnz * sizeof(host.column_indices[0])),
                         hipMemcpyHostToDevice);
  cudaSafeCall(stat);

  stat = hipMemcpy(values, host.values,
    static_cast<std::size_t>(nnz * sizeof(host.values[0])),
                 hipMemcpyHostToDevice);
  cudaSafeCall(stat);

  status = hipsparseXcoo2csr(handle,
    row_indices, nnz, rows, csr_row_ptr, HIPSPARSE_INDEX_BASE_ZERO);
  cusparseSafeCall(status);
}

CUSPARSE::CUSPARSE() : Algorithm() {
}

CUSPARSE::~CUSPARSE() {
}

void CUSPARSE::Init(const reader::Entry& entry) {
  Algorithm::Init(entry);
  internal = new CUSPARSEInternals;

  int N = static_cast<int>(entry.vertices);

  internal->host.A.rows = entry.vertices;
  internal->host.B.rows = entry.vertices;
  internal->host.C.rows = entry.vertices;

  internal->dev.A.rows = entry.vertices;
  internal->dev.B.rows = entry.vertices;
  internal->dev.C.rows = entry.vertices;

  internal->host.A.nnz = 0;
  internal->host.B.nnz = 0;
  internal->host.C.nnz = 0;

  internal->dev.A.nnz = 0;
  internal->dev.B.nnz = 0;
  internal->dev.C.nnz = 0;

  hipsparseStatus_t stat;
  hipError_t status;

  stat = hipsparseCreate(&internal->handle);
  cusparseSafeCall(stat);

  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      if (A[i * N + j] > 1e-10) {
        internal->host.A.nnz++;
      }
      if (B[i * N + j] > 1e-10) {
        internal->host.B.nnz++;
      }
      if (C[i * N + j] > 1e-10) {
        internal->host.C.nnz++;
      }
    }
  }

  // printf("host.C.nnz = %d\n", internal->host.C.nnz);

  internal->host.A.Allocate();
  internal->host.B.Allocate();
  // internal->host.C.Allocate();

  std::size_t ca = 0, cb = 0, cc = 0;  // Current index

  for (std::size_t i = 0; i < N; ++i) {
    for (std::size_t j = 0; j < N; ++j) {
      if (A[i * N + j] - 0.0 > 1e-10) {
        internal->host.A.row_indices[ca] = i;
        internal->host.A.column_indices[ca] = j;
        internal->host.A.values[ca] = A[i * N + j];
        ca++;
      }
      if (B[i * N + j] - 0.0 > 1e-10) {
        internal->host.B.row_indices[cb] = i;
        internal->host.B.column_indices[cb] = j;
        internal->host.B.values[cb] = B[i * N + j];
        cb++;
      }
      // if (C[i * N + j] - 0.0 > 1e-10) {
      //   internal->host.C.row_indices[cc] = i;
      //   internal->host.C.column_indices[cc] = j;
      //   internal->host.C.values[cc] = C[i * N + j];
      //   cc++;
      // }
    }
  }

  internal->dev.A.CopyFrom(internal->host.A, internal->handle);
  internal->dev.B.CopyFrom(internal->host.B, internal->handle);

  stat = hipsparseSetPointerMode(internal->handle, HIPSPARSE_POINTER_MODE_HOST);
  cusparseSafeCall(stat);

  status = hipMalloc((void**)&internal->dev.C.csr_row_ptr,
    (internal->dev.C.rows + 1) * sizeof(internal->dev.C.csr_row_ptr[0]));
  cudaSafeCall(status);

  int baseC = 0, nnzC = 0;
  int *nnzTotalDevHostPtr = &nnzC;

  stat = hipsparseXcsrgemmNnz(internal->handle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              N, N, N,
                              internal->dev.A.mat,
                              internal->dev.A.nnz,
                              internal->dev.A.csr_row_ptr,
                              internal->dev.A.column_indices,
                              internal->dev.B.mat,
                              internal->dev.B.nnz,
                              internal->dev.B.csr_row_ptr,
                              internal->dev.B.column_indices,
                              internal->dev.C.mat,
                              internal->dev.C.csr_row_ptr,
                              nnzTotalDevHostPtr);
  cusparseSafeCall(stat);

  if (nnzTotalDevHostPtr != NULL){
    nnzC = *nnzTotalDevHostPtr;
  } else {
    status = hipMemcpy(&nnzC, internal->dev.C.csr_row_ptr+N, sizeof(int),
               hipMemcpyDeviceToHost);
    cudaSafeCall(status);
    status = hipMemcpy(&baseC, internal->dev.C.csr_row_ptr, sizeof(int),
               hipMemcpyDeviceToHost);
    cudaSafeCall(status);
    nnzC -= baseC;
  }
  internal->host.C.nnz = nnzC;
  internal->dev.C.nnz = nnzC;

  status = hipMalloc((void**)&internal->dev.C.values,
    internal->dev.C.nnz * sizeof(internal->dev.C.values[0]));
  cudaSafeCall(status);

  status = hipMalloc((void**)&internal->dev.C.column_indices,
    internal->dev.C.nnz * sizeof(internal->dev.C.column_indices[0]));
  cudaSafeCall(status);

  // internal->dev.C.CopyFrom(internal->host.C, internal->handle);
}

void CUSPARSE::Finalize() {
  hipsparseDestroy(internal->handle);
  delete internal;
}

void CUSPARSE::Compute() {
  int N = static_cast<int>(entry.vertices);
  hipsparseStatus_t status;

  status = hipsparseScsrgemm(internal->handle,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                    N, N, N,
                    internal->dev.A.mat,
                      internal->dev.A.nnz,
                      internal->dev.A.values,
                      internal->dev.A.csr_row_ptr,
                      internal->dev.A.column_indices,
                    internal->dev.B.mat,
                      internal->dev.B.nnz,
                      internal->dev.B.values,
                      internal->dev.B.csr_row_ptr,
                      internal->dev.B.column_indices,
                    internal->dev.C.mat,
                      internal->dev.C.values,
                      internal->dev.C.csr_row_ptr,
                      internal->dev.C.column_indices);
  cusparseSafeCall(status);
}

} // namespace core

